#include "hip/hip_runtime.h"
//Cuda file 
#include "hip/hip_runtime.h"  
#include "" 
#include <opencv2/core/core.hpp>  
#include <opencv2/highgui/highgui.hpp>  
#include <opencv2/imgproc/types_c.h>  
#include <opencv2/imgproc/imgproc.hpp>  



using namespace cv; 

__global__ void Kernel_EraseBackground(unsigned char *MatA, unsigned char *MatR, int rows, int cols, uchar b1, uchar b2, uchar b3, bool fChange, uchar f1, uchar f2, uchar f3, int min, int max);
__global__ void Kernel_SobelFilter(unsigned char *MatA, unsigned char *MatR, int rows, int cols, double mult);

__device__  double Min(double red, double blue, double green); 
__device__  double Max(double red, double blue, double green);
__device__ int sobel(int a, int b, int c, int d, int e, int f);

int iDivUp(int a, int b);
int iAlignUp(int a, int b);

extern "C" void ApplySobelFilter(Mat *ptMatA, Mat *ptMatR, double mult)
{
	hipError_t error;
	//pointeurs des matrices 
	uchar *MatA, *MatR;
	//Dimension de la grid et des blocs 
	dim3 nbreThreadsParBlock(32, 32);
	dim3 nbreBloc(iDivUp(ptMatA->cols, 32), iDivUp(ptMatA->rows, 32));

	//Allouer espace pour le gpu 
	int memSize = ptMatA->rows * ptMatA->step1();

	hipMalloc((void **)&MatA, memSize);
	hipMalloc((void **)&MatR, memSize);

	//Envoyer matrice dans la m�moire du gpu 
	hipMemcpy(MatA, ptMatA->data, memSize, hipMemcpyHostToDevice);
	Kernel_SobelFilter << <nbreBloc, nbreThreadsParBlock >> >(MatA, MatR, ptMatA->step1(), ptMatA->rows, mult);
	//Wait the Kernel to be done
	hipDeviceSynchronize();
	//Retourner la matrice r�sultante 
	hipMemcpy(ptMatR->data, MatR, memSize, hipMemcpyDeviceToHost);
	error = hipFree(MatA);
	error = hipFree(MatR);
}
extern "C" void EraseBackground(Mat *ptMatA, Mat *ptMatR, uchar backgroundColor[], uchar foregroundColor[], int threshMin, int threshMax)
{ 
	hipError_t error;
	//pointeurs des matrices 
	uchar *MatA, *MatR;
	//Dimension de la grid et des blocs 
	dim3 nbreThreadsParBlock(32, 32);
	dim3 nbreBloc(iDivUp(ptMatA->cols, 32), iDivUp(ptMatA->rows, 32));

	//Allouer espace pour le gpu 
	int memSize = ptMatA->rows * ptMatA->step1();

	hipMalloc((void **)&MatA, memSize);
	hipMalloc((void **)&MatR, memSize);

	//Envoyer matrice dans la m�moire du gpu 
	hipMemcpy(MatA, ptMatA->data, memSize, hipMemcpyHostToDevice);
	if (foregroundColor != NULL) 
	{
		Kernel_EraseBackground << <nbreBloc, nbreThreadsParBlock >> >(MatA, MatR, ptMatA->step1(), ptMatA->rows, backgroundColor[0], backgroundColor[1], backgroundColor[2],
			true, foregroundColor[0], foregroundColor[1], foregroundColor[2], threshMin, threshMax);
	}
	else 
	{
		Kernel_EraseBackground << <nbreBloc, nbreThreadsParBlock >> >(MatA, MatR, ptMatA->step1(), ptMatA->rows, backgroundColor[0], backgroundColor[1], backgroundColor[2],
			false, 0, 0, 0, threshMin, threshMax);

	}
	
	//Wait the Kernel to be done
	hipDeviceSynchronize();
	//Retourner la matrice r�sultante 
	hipMemcpy(ptMatR->data, MatR, memSize, hipMemcpyDeviceToHost);
	error = hipFree(MatA);
	error = hipFree(MatR);
	
} 
 
__global__ void Kernel_SobelFilter(unsigned char *MatA, unsigned char *MatR, int rows, int cols, double mult)
{
	//GradiantX
	int kernelx[3][3] = { { -1, 0, 1 },
							{ -2, 0, 2 },
							{ -1, 0, 1 } };

	//GradiantY
	int kernely[3][3] = { { -1, -2, -1 },
							{ 0,  0,  0 },
							{ 1,  2,  1 } };
	//X et Y dans la matrice 
	int ImgNumColonne = (blockIdx.x  * blockDim.x) + threadIdx.x;
	int ImgNumLigne = (blockIdx.y * blockDim.y) + threadIdx.y;
	int Index = (ImgNumLigne * rows) + (ImgNumColonne * 3);

	//Ne depasse pas l'acc�s de la matrice
	if ((ImgNumColonne < (rows) - 1) && (ImgNumLigne < (cols) -2)) 
	{
		
		//Emplacement dans la m�moire
		int x1 = Index;
		int x2 = Index + 1;
		int x3 = Index + 2;
		int x4 = ((ImgNumLigne + 1) * rows) + ((ImgNumColonne) * 3);
		int x5 = ((ImgNumLigne + 1) * rows) + ((ImgNumColonne ) * 3) + 1;
		int x6 = ((ImgNumLigne + 1) * rows) + ((ImgNumColonne)* 3) + 2;
		int x7 = ((ImgNumLigne + 2) * rows) + ((ImgNumColonne) * 3);
		int x8 = ((ImgNumLigne + 2) * rows) + ((ImgNumColonne)* 3) + 1;
		int x9 = ((ImgNumLigne + 2) * rows) + ((ImgNumColonne) * 3) + 2;

		int magX = (kernelx[0][0] * MatA[x1]) + (kernelx[0][1] * MatA[x2]) + (kernelx[0][2] * MatA[x3]) +
			(kernelx[1][0] * MatA[x4]) + (kernelx[1][1] * MatA[x5]) + (kernelx[1][2] * MatA[x6]) +
			(kernelx[2][0] * MatA[x7]) + (kernelx[2][1] * MatA[x8]) + (kernelx[2][2] * MatA[x9]);

		int magY = (kernely[0][0] * MatA[x1]) + (kernely[0][1] * MatA[x2]) + (kernely[0][2] * MatA[x3]) +
			(kernely[1][0] * MatA[x4]) + (kernely[1][1] * MatA[x5]) + (kernely[1][2] * MatA[x6]) +
			(kernely[2][0] * MatA[x7]) + (kernely[2][1] * MatA[x8]) + (kernely[2][2] * (MatA[x9]));

		

		int magT = hypotf(magX, magY)*mult;
		MatR[x5] = (uchar)magT;
	}

}
__global__ void Kernel_EraseBackground(unsigned char *MatA, unsigned char *MatR, int rows, int cols, uchar b1, uchar b2, uchar b3,bool fChange, uchar f1, uchar f2, uchar f3, int min, int max)
{ 
	//X et Y dans la matrice 
	int ImgNumColonne = (blockIdx.x  * blockDim.x) + threadIdx.x; 
	int ImgNumLigne = (blockIdx.y * blockDim.y) + threadIdx.y; 
	int Index = (ImgNumLigne * rows)  + (ImgNumColonne * 3);

	if ((ImgNumColonne < rows / 3) && (ImgNumLigne < cols))
	{
		double blue = (double)MatA[Index] / 255;
		double green = (double)MatA[Index + 1] / 255;
		double red = (double)MatA[Index + 2] / 255;

		double cMax = Max(red, blue, green);

		double cMin = Min(red, blue, green);

		double delta = cMax - cMin;

		//	HUE
		double h = 0;
		if (blue == cMax) {
			h = 60 * ((red - green) / delta + 4);
		}
		else if (green == cMax) {
			h = 60 * ((blue - red) / delta + 2);
		}
		else if (red == cMax) {
			h = 60 * ((green - blue) / delta);
			if (h < 0)
				h += 360;
		}

		//	SATURATION
		double saturation = 0;
		if (cMax != 0) {
			saturation = delta / cMax;
		}

		//	VALUE
		double value = cMax;

		if (h / 2 > min && h / 2 < max)
		{
			MatR[Index] = b1;
			MatR[Index + 1] = b2;
			MatR[Index + 2] = b3;
		}
		else 
		{
			if (fChange == true)
			{
				MatR[Index] = f1;
				MatR[Index + 1] = f2;
				MatR[Index + 2] = f3;
			}
			else 
			{
				MatR[Index] = h/2;
				MatR[Index + 1] = saturation*255;
				MatR[Index + 2] = value*255;
			}
		}
	}

	
	
	return; 
 
} 

__device__  double Min(double red, double blue, double green)
{ 
	if ((blue <= green) && (blue <= red))
		return blue;
	else if ((green <= blue) && (green <= red))
		return green;
	else
		return red;
} 
__device__  double Max(double red, double blue, double green)
{ 
	if ((blue >= green) && (blue >= red))
		return blue;
	else if ((green >= blue) && (green >= red))
		return green;
	else
		return red;
} 

__device__ int sobel(int a, int b, int c, int d, int e, int f) {
	return ((a + 2 * b + c) - (d + 2 * e + f));
}

int iDivUp(int a, int b) // Round a / b to nearest higher integer value

{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}
int iAlignUp(int a, int b) // Align a to nearest higher multiple of b

{
	return (a % b != 0) ? (a - a % b + b) : a;
}

 
 